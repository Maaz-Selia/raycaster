#include "hip/hip_runtime.h"
/*
*   Built upon bicubicTexture sample and RT Lab
*/

    #pragma region includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <sphere.h>
#include <hitable.h>
#include <hitable_list.h>
#include <vec3.h>
#include <Ray.h>
    #pragma endregion

typedef unsigned int uint;
typedef unsigned char uchar;


    #pragma region OpenGL

hipArray* d_imageArray = 0;

extern "C"
void initTexture(int imageWidth, int imageHeight, uchar * h_data)
{
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_imageArray;
}

extern "C"
void freeTexture()
{
    checkCudaErrors(hipFreeArray(d_imageArray));
}

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

    #pragma endregion

    #pragma region Random Number Generator
// Random Number generation
// http://ianfinlayson.net/class/cpsc425/notes/cuda-random
# define MAX_RANDOM 99
__device__ void random(int* result, int seed) {
    __shared__ hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    *result = hiprand(&state) % MAX_RANDOM;
}
    #pragma endregion

# define PARTICLE_COUNT 20

static hitable** d_list;
static hitable** d_world;

__device__ vec3 castRay(const ray& r, hitable** world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec))
    {
        return 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    }
    else
    {
        return vec3(0, 0, 0);
    }
}

__global__ void free_world(hitable** d_list, hitable** d_world)
{
    delete* (d_list);
    delete* (d_list + 1);
    delete* d_world;
}

__global__ void create_world(hitable** d_list, hitable** d_world)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {

        int randomNumbers[PARTICLE_COUNT * 6];

        for (int i = 0; i < PARTICLE_COUNT * 6; i++) {
            int* num = &randomNumbers[i];
            random(num, i+1);
            //printf("%d\n", randomNumbers[i]);
        }

        for (int i = 0; i < PARTICLE_COUNT; i++) {
            int randomIndex = i * 6;
            vec3 loc = vec3(randomNumbers[randomIndex] - 50, randomNumbers[randomIndex + 1]- 50, randomNumbers[randomIndex + 2] * -1);
            vec3 vel = vec3(randomNumbers[randomIndex + 3] - 50, randomNumbers[randomIndex + 4] - 50, randomNumbers[randomIndex + 5] -50)/10;
            //printf("%d", randomNumbers[i]);
            *(d_list + i) = new sphere(loc, vel);
        }

        *d_world = new hitable_list(d_list, PARTICLE_COUNT);
    }
}

//
//__device__ vec3 collisionReVel(sphere particle) {
//    vec3 a = particle.vel - vel;
//    vec3 iniMom = particle.vel + vel;
//    vec3 v2After = (iniMom - a) / 2;
//    return v2After + a;
//}

__global__ void d_gravity(hitable** world) {
    (*world)->applyGravity();
}

__global__ void d_letChaos(hitable** world) {
    __shared__ int randomNumbers[PARTICLE_COUNT * 3];

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < PARTICLE_COUNT * 3; i++) {
            int* num = &randomNumbers[i];
            random(num, i + 1);
        }
    }
    __syncthreads();

    int randomIndex = threadIdx.x * 3;
    vec3 newVel = vec3(randomNumbers[randomIndex] - 50, randomNumbers[randomIndex + 1] - 50, randomNumbers[randomIndex + 2] - 50) / 10;
    (*world)->letChaos(threadIdx.x, newVel);
}

__global__ void d_moveParticles(hitable** world) {
    (*world)->reVel(threadIdx.x);
    (*world)->move(threadIdx.x);
}

__global__ void d_raycast(uchar4* d_output, uint width, uint height, hitable** d_world)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint i = y * width + x;

    float u = float(x) / float(width);
    float v = float(y) / float(height);

    u = 2.0 * u - 1;
    v = -(2.0 * v - 1);

    u *= float(width) / float(height);

    u *= 2.0;
    v *= 2.0;

    vec3 eye = vec3(0, 0, 1.5);
    float distFrEye2Img = 1.0;

    if ((x < width) && (y < height))
    {
        //for each pixel
        vec3 pixelPos = vec3(u, v, eye.z() - distFrEye2Img);
        //fire a ray:
        ray r;
        r.O = eye;

        r.Dir = pixelPos - eye;    //view direction along negtive z-axis
        vec3 col = castRay(r, d_world);
        float red = col.x();
        float green = col.y();
        float blue = col.z();
        d_output[i] = make_uchar4(red * 255, green * 255, blue * 255, 0);
    }
}


// render image using CUDA
extern "C"
void render(int width, int height, dim3 blockSize, dim3 gridSize, uchar4 * output)
{
    _sleep(10);
    d_moveParticles << <1, PARTICLE_COUNT >> > (d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    d_raycast << <gridSize, blockSize >> > (output, width, height, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

extern "C"
void applyGravity() {
    checkCudaErrors(hipDeviceSynchronize());
    d_gravity << <1, PARTICLE_COUNT >> > (d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

extern "C"
void letChaos() {
    checkCudaErrors(hipDeviceSynchronize());
    d_letChaos << <1, PARTICLE_COUNT >> > (d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

extern "C"
void particlesInitialise(int width, int height, dim3 blockSize, dim3 gridSize)
{
    checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(hitable*)));
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));

    
    create_world << <1, 1>> > (d_list, d_world);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

